
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <omp.h>
#include <chrono>
#include <ctime>
#define INF 1073741823
#define BS 64
#define BS_DIM 32

using namespace std;

__device__ void floyd(int C[][BS], int A[][BS], int B[][BS], int ti, int tj) {
    int sum;
    for(int k=0; k<BS; k++) {
        for(int i=0; i<BS; i+=BS_DIM) {
            for(int j=0; j<BS; j+=BS_DIM) {
                sum = A[ti+i][k] + B[k][tj+j];
                if(C[ti+i][tj+j] > sum) {
                    C[ti+i][tj+j] = sum;
                }
            }
        }
        __syncthreads();
    }
}

//Wkk, Wkk, Wkk
__global__ void floyd_warshall_phase1(int* mat, int V, int k) {
    __shared__ int C[BS][BS];
    int ti = threadIdx.y;
    int tj = threadIdx.x;
    int kti = k*BS+ti;
    int ktj = k*BS+tj;
    for(int i=0; i<BS; i+=BS_DIM) {
        for(int j=0; j<BS; j+=BS_DIM) {
            C[ti+i][tj+j] = mat[(kti+i)*V+ktj+j];
        }
    }
    
    __syncthreads();
    floyd(C, C, C, ti, tj);
    for(int i=0; i<BS; i+=BS_DIM) {
        for(int j=0; j<BS; j+=BS_DIM) {
            mat[(kti+i)*V+ktj+j] = C[ti+i][tj+j];
        }
    }
}

//Wkj, Wkk, Wkj
//Wik, Wik, Wkk
__global__ void floyd_warshall_phase2(int* mat, int V, int k) {
    __shared__ int C[BS][BS];
    __shared__ int A[BS][BS];
    int bidx = blockIdx.x;
    int bidy = blockIdx.y;
    int ti = threadIdx.y;
    int tj = threadIdx.x;
    if(bidx==k) return;
    for(int i=0; i<BS; i+=BS_DIM) {
        for(int j=0; j<BS; j+=BS_DIM) {
            A[ti+i][tj+j] = mat[(k*BS+ti+i)*V+k*BS+tj+j];
        }
    }
    int base_i, base_j;
    if(bidy==0) {
        base_i = k*BS+ti;
        base_j = bidx*BS+tj;
        for(int i=0; i<BS; i+=BS_DIM) {
            for(int j=0; j<BS; j+=BS_DIM) {
                C[ti+i][tj+j] = mat[(k*BS+ti+i)*V+bidx*BS+tj+j];
            }
        }
        __syncthreads();
        floyd(C, A, C, ti, tj);
    }
    else {
        base_i = bidx*BS+ti;
        base_j = k*BS+tj;
        for(int i=0; i<BS; i+=BS_DIM) {
            for(int j=0; j<BS; j+=BS_DIM) {
                C[ti+i][tj+j] = mat[(bidx*BS+ti+i)*V+k*BS+tj+j];
            }
        }
        __syncthreads();
        floyd(C, C, A, ti, tj);
    }

    mat[base_i*V+base_j] = C[ti][tj];
    mat[base_i*V+base_j+BS_DIM] = C[ti][tj+BS_DIM];
    mat[(base_i+BS_DIM)*V+base_j] = C[ti+BS_DIM][tj];
    mat[(base_i+BS_DIM)*V+base_j+BS_DIM] = C[ti+BS_DIM][tj+BS_DIM];
}
//Wij, Wik, Wkj
__global__ void floyd_warshall_phase3(int* mat, int V, int k) {
    __shared__ int A[BS][BS];
    __shared__ int B[BS][BS];
    // __shared__ int C[BS][BS];
    int bi = blockIdx.y;
    int bj = blockIdx.x;
    int ti = threadIdx.y;
    int tj = threadIdx.x;
    if(bj==k || bi==k) return;

    int biti = bi*BS+ti;
    int bjtj = bj*BS+tj;
    int kti  = k*BS+ti;
    int ktj  = k*BS+tj;

    for(int i=0; i<BS; i+=BS_DIM) {
        for(int j=0; j<BS; j+=BS_DIM) {
            A[ti+i][tj+j] = mat[(biti+i)*V+ktj+j];
        }
    }
    for(int i=0; i<BS; i+=BS_DIM) {
        for(int j=0; j<BS; j+=BS_DIM) {
            B[ti+i][tj+j] = mat[(kti+i)*V+bjtj+j];
        }
    }
    __syncthreads();
    int i0, i1, i2, i3;
    i0 = mat[biti*V+bjtj];
    i1 = mat[biti*V+bjtj+BS_DIM];
    i2 = mat[(biti+BS_DIM)*V+bjtj];
    i3 = mat[(biti+BS_DIM)*V+bjtj+BS_DIM];

    for(int kk=0; kk<BS; kk++) {
        int sum0 = A[ti][kk] + B[kk][tj];
        int sum1 = A[ti][kk] + B[kk][tj+BS_DIM];
        int sum2 = A[ti+BS_DIM][kk] + B[kk][tj];
        int sum3 = A[ti+BS_DIM][kk] + B[kk][tj+BS_DIM];
        if(i0 > sum0) {
            i0 = sum0;
        }
        if(i1 > sum1) {
            i1 = sum1;
        }
        if(i2 > sum2) {
            i2 = sum2;
        }
        if(i3 > sum3) {
            i3 = sum3;
        }
    }

    mat[biti*V + bjtj]                 = i0;
    mat[biti*V + bjtj+BS_DIM]          = i1;
    mat[(biti+BS_DIM)*V + bjtj]        = i2;
    mat[(biti+BS_DIM)*V + bjtj+BS_DIM] = i3;
}

int main(int argc, char** argv)
{
    auto start_time = chrono::steady_clock::now();
    int numOfVertex;
    int numOfEdge;
    ifstream in;
    ofstream out;
    in.open(argv[1], ios::binary | ios::in);
    in.read((char*)&numOfVertex, sizeof(int));
    in.read((char*)&numOfEdge, sizeof(int));

    const int int_size = sizeof(int);
    // padding
    const int size = ((numOfVertex+BS-1)/BS)*BS;
    int* mat = new int[size*size];
    for(int i=0; i<size; i++) {
        for(int j=0; j<size; j++) {
            if(i>=numOfVertex || j>=numOfVertex) mat[i*size+j] = INF;
            else if(i==j) mat[i*size+j] = 0;
            else mat[i*size+j] = INF;
        }
    }
    // auto start_time = chrono::steady_clock::now();
    for(int i=0; i<numOfEdge; i++) {
        int src, dest;
        in.read((char*)&src, int_size);
        in.read((char*)&dest, int_size);
        in.read((char*)&mat[src*size+dest], int_size);
    }
    auto end_time = chrono::steady_clock::now();
    auto time_span = chrono::duration_cast<chrono::duration<double>>(end_time - start_time);
    cout << "read time: " << time_span.count() << endl;

    size_t mat_size = sizeof(int) * size * size;
    int* device_mat;
    hipMalloc(&device_mat, mat_size);

    start_time = chrono::steady_clock::now();
    hipMemcpy(device_mat, mat, mat_size, hipMemcpyHostToDevice);
    end_time = chrono::steady_clock::now();
    time_span = chrono::duration_cast<chrono::duration<double>>(end_time - start_time);
    cout << "copy time: " << time_span.count() << endl;

    start_time = chrono::steady_clock::now();
    int numOfBlock = size/BS;
    dim3 block_dim(BS_DIM, BS_DIM, 1);
    dim3 p2_grid(numOfBlock, 2, 1);
    dim3 grid_dim(numOfBlock, numOfBlock, 1);

    for (int k=0; k<numOfBlock; k++) {
        floyd_warshall_phase1<<<1, block_dim>>>(device_mat, size, k);
        floyd_warshall_phase2<<<p2_grid, block_dim>>>(device_mat, size, k);
        floyd_warshall_phase3<<<grid_dim, block_dim>>>(device_mat, size, k);
    }
    hipDeviceSynchronize();

    end_time = chrono::steady_clock::now();
    time_span = chrono::duration_cast<chrono::duration<double>>(end_time - start_time);
    cout << "computation time: " << time_span.count() << endl;

    start_time = chrono::steady_clock::now();
    hipMemcpy(mat, device_mat, mat_size, hipMemcpyDeviceToHost);
    end_time = chrono::steady_clock::now();
    time_span = chrono::duration_cast<chrono::duration<double>>(end_time - start_time);
    cout << "copy time: " << time_span.count() << endl;

    hipFree(device_mat);

    hipError_t err = hipGetLastError();

    if( err != hipSuccess ) {
        printf("CUDA Error: %s\n", hipGetErrorString(err)); 
    }
    // print(mat, numOfVertex);
    
    start_time = chrono::steady_clock::now();
    out.open(argv[2], ios::binary | ios::out);
    for(int i=0; i<numOfVertex; i++) {
        out.write((char*)&mat[i*size], numOfVertex*sizeof(int));
    }
    end_time = chrono::steady_clock::now();
    time_span = chrono::duration_cast<chrono::duration<double>>(end_time - start_time);
    cout << "write time: " << time_span.count() << endl;


    return 0;
}